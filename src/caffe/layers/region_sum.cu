#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {


template <typename Dtype>
__global__ void forward_kernel(const int n, const Dtype* data1, const Dtype* data2, const Dtype* data_mask,
    const int spatial_dim, const Dtype op1_, const Dtype op2_, const int mask_cnt, Dtype* data) {
  CUDA_KERNEL_LOOP(index, n) {
    const int temp = static_cast<int>(data_mask[index % spatial_dim]);
    data[index] = data1[index] * op1_ + ((temp == -1) ? 0 : data2[(index / spatial_dim) * mask_cnt + temp] * op2_);
  }
}

template <typename Dtype>
void RegionSumLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

  const Dtype* bottom_data0 = bottom[0]->gpu_data();
  const Dtype* bottom_data1 = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* mask_data = bottom[2]->gpu_data();
  const int spatial_dim = top[0]->height() * top[0]->width();
  const int count = spatial_dim * top[0]->channels();
  

  for (int i=0; i<bottom[0]->num(); i++)
  {
    int mask_cnt = static_cast<int>(bottom[3]->cpu_data()[i]);
    forward_kernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data0, bottom_data1, mask_data, spatial_dim, op1_, op2_, mask_cnt,
        top_data);

    bottom_data0 += bottom[0]->offset(1);
    bottom_data1 += bottom[1]->offset(1);
    mask_data += bottom[2]->offset(1);
    top_data += top[0]->offset(1);
  }
}

template <typename Dtype>
__global__ void backward_kernel(const int n, Dtype* data1, Dtype* data2, const Dtype* data_mask,
    const int spatial_dim, const Dtype op1_, const Dtype op2_, const int mask_cnt, const Dtype* data) {
  CUDA_KERNEL_LOOP(index, n) {
    data1[index] = data[index] * op1_;
    const int temp = static_cast<int>(data_mask[index % spatial_dim]);
    if (temp!=-1)
      data2[(index / spatial_dim) * mask_cnt + temp] = data[index] * op2_;
  }
}



template <typename Dtype>
void RegionSumLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  Dtype* bottom_data0 = bottom[0]->mutable_gpu_diff();
  Dtype* bottom_data1 = bottom[1]->mutable_gpu_diff();
  const Dtype* top_data = top[0]->gpu_diff();
  const Dtype* mask_data = bottom[2]->gpu_data();
  const int spatial_dim = top[0]->height() * top[0]->width();
  const int count = spatial_dim * top[0]->channels();


  for (int i=0; i<bottom[0]->num(); i++)
  {
    int mask_cnt = static_cast<int>(bottom[3]->cpu_data()[i]);
    backward_kernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data0, bottom_data1, mask_data, spatial_dim, op1_, op2_, mask_cnt,
        top_data);

    bottom_data0 += bottom[0]->offset(1);
    bottom_data1 += bottom[1]->offset(1);
    mask_data += bottom[2]->offset(1);
    top_data += top[0]->offset(1);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(RegionSumLayer);

}  // namespace caffe
