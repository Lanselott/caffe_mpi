#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/region_im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void move_back_kernel(const int n, const Dtype* data_mask, const Dtype* top_buffer,
    const int spatial_dim, const int mask_cnt, Dtype* data) {
  CUDA_KERNEL_LOOP(index, n) {
    const int temp = static_cast<int>(data_mask[index % spatial_dim]);
    data[index] = (temp == -1) ? 0 : top_buffer[(index / spatial_dim) * mask_cnt + temp];
  }
}

template <typename Dtype>
__global__ void compression_move_back_kernel(const int n, const Dtype* top_buffer, Dtype* data) {
  CUDA_KERNEL_LOOP(index, n) {
    data[index] = top_buffer[index];
  }
}

template <typename Dtype>
void RegionConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  
  const Dtype* weights = this->blobs_[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* top_buffer = top_buffer_->mutable_gpu_data();
  const Dtype* mask_data = bottom[1]->gpu_data();
  const Dtype* index_1 = bottom[2]->gpu_data()+bottom[2]->offset(0, 0, 0, 1);
  const Dtype* index_2 = bottom[2]->gpu_data()+bottom[2]->offset(0, 0, 1, 1);
  const int count = top[0]->count();
  int mask_cnt_ = bottom[2]->cpu_data()[0];

  if (mask_cnt_!=0)
  {
    //region im2col
    if (!input_compression_)
    {
      region_im2col_gpu(bottom_data, index_1, index_2, mask_cnt_, conv_in_channels_, conv_in_height_, conv_in_width_,
            kernel_h_, kernel_w_, pad_h_, pad_w_, dilation_h_, dilation_w_, col_buffer_->mutable_gpu_data());
    }
    else
    {
      compression_region_im2col_gpu(bottom_data, bottom[1]->gpu_data(), index_1, index_2, mask_cnt_, conv_in_channels_, conv_in_height_, conv_in_width_,
            kernel_h_, kernel_w_, pad_h_, pad_w_, dilation_h_, dilation_w_, col_buffer_->mutable_gpu_data());
    }

    //gemmm
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, conv_out_channels_, mask_cnt_, kernel_dim_,
        (Dtype)1., weights, col_buffer_->gpu_data(),
        (Dtype)0., top_buffer);

    //bias
    if (this->bias_term_) {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_output_,
      mask_cnt_, 1, (Dtype)1., this->blobs_[1]->gpu_data(), bias_multiplier_.gpu_data(),
      (Dtype)1., top_buffer);
    }
  }
  //move back
  //caffe_gpu_set(count, static_cast<Dtype>(0), top_data);
  if (!output_compression_)
  {
    move_back_kernel<Dtype><<<CAFFE_GET_BLOCKS(conv_out_spatial_dim_ * conv_out_channels_), CAFFE_CUDA_NUM_THREADS>>>(
          conv_out_spatial_dim_ * conv_out_channels_, mask_data, top_buffer_->gpu_data(), conv_out_spatial_dim_, mask_cnt_,
          top_data);
  }
  else
  {
    compression_move_back_kernel<Dtype><<<CAFFE_GET_BLOCKS(mask_cnt_ * conv_out_channels_), CAFFE_CUDA_NUM_THREADS>>>(
        mask_cnt_ * conv_out_channels_, top_buffer_->gpu_data(), top_data);
  }
  CUDA_POST_KERNEL_CHECK;
}



template <typename Dtype>
__global__ void pick_out_kernel(const int n, const Dtype* data_diff,
    const int height, const int width,
    const Dtype* index_1, const Dtype* index_2,
    const int mask_cnt, Dtype* diff_buffer) {
  CUDA_KERNEL_LOOP(index, n) {
    const int m_index = index % mask_cnt;
    const int c = index / mask_cnt;
    const int h = index_1[m_index];
    const int w = index_2[m_index];
    diff_buffer[index] = data_diff[(c * height + h) * width + w];
  }
}

template <typename Dtype>
__global__ void compression_pick_out_kernel(const int n, const Dtype* data_diff, Dtype* diff_buffer) {
  CUDA_KERNEL_LOOP(index, n) {
    diff_buffer[index] = data_diff[index];
  }
}



template <typename Dtype>
void RegionConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  const Dtype* weights = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();

  const Dtype* top_diff = top[0]->gpu_diff();

  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

  const Dtype* mask_data = bottom[1]->gpu_data();
  const Dtype* index_1 = bottom[2]->gpu_data()+bottom[2]->offset(0, 0, 0, 1);
  const Dtype* index_2 = bottom[2]->gpu_data()+bottom[2]->offset(0, 0, 1, 1);
  const int count = top[0]->count();
  int mask_cnt_ = bottom[2]->cpu_data()[0];

  //pick_out_kernel
  int num_kernels = conv_out_channels_ * mask_cnt_;

  if (!output_compression_)
  {
    pick_out_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
          num_kernels, top_diff, height_out_, width_out_, index_1, index_2, mask_cnt_, top_buffer_->mutable_gpu_diff());
  }
  else
  {
    compression_pick_out_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, top_diff, top_buffer_->mutable_gpu_diff());
  }


  // Bias gradient, if necessary.
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    caffe_gpu_gemv<Dtype>(CblasNoTrans, num_output_, mask_cnt_, 1.,
        top_buffer_->gpu_diff(), bias_multiplier_.gpu_data(), 1., this->blobs_[1]->mutable_gpu_diff());
  }

  // weight gradient
  if (this->param_propagate_down_[0]) {

    if (!input_compression_)
    {
      region_im2col_gpu(bottom_data, index_1, index_2, mask_cnt_, conv_in_channels_, conv_in_height_, conv_in_width_,
            kernel_h_, kernel_w_, pad_h_, pad_w_, dilation_h_, dilation_w_, col_buffer_->mutable_gpu_data());
    }
    else
    {
      compression_region_im2col_gpu(bottom_data, bottom[1]->gpu_data(), index_1, index_2, mask_cnt_, conv_in_channels_, conv_in_height_, conv_in_width_,
            kernel_h_, kernel_w_, pad_h_, pad_w_, dilation_h_, dilation_w_, col_buffer_->mutable_gpu_data());
    }

    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, conv_out_channels_,
        kernel_dim_, mask_cnt_,
        (Dtype)1., top_buffer_->gpu_diff() , col_buffer_->gpu_data(),
        (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
  }

  //data gradient
  if (propagate_down[0]) {
    caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, kernel_dim_,
        mask_cnt_, conv_out_channels_,
        (Dtype)1., weights , top_buffer_->gpu_diff(),
        (Dtype)0., col_buffer_->mutable_gpu_data());
    caffe_gpu_set(bottom[0]->count(), static_cast<Dtype>(0), bottom_diff);
    if (!input_compression_)
    {
      region_col2im_gpu(col_buffer_->gpu_data(), 
          index_1, index_2, mask_data,
          mask_cnt_, conv_in_channels_,
          conv_in_height_, conv_in_width_, kernel_h_, kernel_w_,
          pad_h_, pad_w_, dilation_h_, dilation_w_,
          bottom_diff);
    }
    else
    {
      compression_region_col2im_gpu(col_buffer_->gpu_data(), 
          index_1, index_2, mask_data,
          mask_cnt_, conv_in_channels_,
          conv_in_height_, conv_in_width_, kernel_h_, kernel_w_,
          pad_h_, pad_w_, dilation_h_, dilation_w_,
          bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(RegionConvolutionLayer);

}  // namespace caffe
