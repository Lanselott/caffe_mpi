#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/common_layers.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void BNLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
  const vector<Blob<Dtype>*>& top) {
  const Dtype* const_bottom_data = bottom[0]->gpu_data();
  const Dtype* const_top_data = top[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  const Dtype* scale_data = this->blobs_[0]->gpu_data();
  const Dtype* shift_data = this->blobs_[1]->gpu_data();

  update_max_rd();

  // Mean normalization
  if (frozen_ || this->phase_ == TEST) {
    // Use the moving average mean
    caffe_copy(batch_statistic_.count(), this->blobs_[2]->gpu_data(),
        batch_statistic_.mutable_gpu_data());
  } else {
    // Compute the mean by averaging over spatial and batch dimensions.
    caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * channels_, height_ * width_,
        Dtype(1) / (height_ * width_), const_bottom_data,
        spatial_sum_multiplier_.gpu_data(), Dtype(0),
        spatial_statistic_.mutable_gpu_data());
    caffe_gpu_gemv<Dtype>(CblasTrans, num_, channels_,
        Dtype(1) / num_, spatial_statistic_.gpu_data(),
        batch_sum_multiplier_.gpu_data(), Dtype(0),
        batch_statistic_.mutable_gpu_data());

    // Add to the moving average
    if (!frozen_) {
      caffe_gpu_axpby(batch_statistic_.count(),
          Dtype(1) - bn_momentum_, batch_statistic_.gpu_data(),
          bn_momentum_, this->blobs_[2]->mutable_gpu_data());
    }
    if (this->rebn_)
      caffe_copy(channels_, batch_statistic_.gpu_data(), this->d_.mutable_gpu_data()); // temp buffer
  }
  // Broadcast the mean vector
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1,
      Dtype(1), batch_sum_multiplier_.gpu_data(), batch_statistic_.gpu_data(),
      Dtype(0), spatial_statistic_.mutable_gpu_data());
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_,
      height_ * width_, 1, Dtype(-1),
      spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(),
      Dtype(0), broadcast_buffer_.mutable_gpu_data());
  // Subtract
  caffe_gpu_add(broadcast_buffer_.count(), const_bottom_data,
      broadcast_buffer_.gpu_data(), top_data);

  // Variance normalization
  if (frozen_ || this->phase_ == TEST) {
    // Use the moving average variance
    caffe_copy(batch_statistic_.count(), this->blobs_[3]->gpu_data(),
        batch_statistic_.mutable_gpu_data());
  } else {
    caffe_gpu_powx(broadcast_buffer_.count(), const_top_data, Dtype(2),
        broadcast_buffer_.mutable_gpu_data());
    caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * channels_, height_ * width_,
        Dtype(1) / (height_ * width_), broadcast_buffer_.gpu_data(),
        spatial_sum_multiplier_.gpu_data(), Dtype(0),
        spatial_statistic_.mutable_gpu_data());
    caffe_gpu_gemv<Dtype>(CblasTrans, num_, channels_, Dtype(1) / num_,
        spatial_statistic_.gpu_data(), batch_sum_multiplier_.gpu_data(),
        Dtype(0), batch_statistic_.mutable_gpu_data());

    // Add to the moving average
    caffe_gpu_axpby(batch_statistic_.count(),
        Dtype(1) - bn_momentum_, batch_statistic_.gpu_data(),
        bn_momentum_, this->blobs_[3]->mutable_gpu_data());
    if (this->rebn_)
      caffe_copy(channels_, batch_statistic_.gpu_data(), this->r_.mutable_gpu_data()); // temp buffer
  }

  // Add eps
  caffe_gpu_add_scalar(batch_statistic_.count(), bn_eps_,
        batch_statistic_.mutable_gpu_data());
  // Inverse standard deviation
  caffe_gpu_powx(batch_statistic_.count(), batch_statistic_.gpu_data(),
        Dtype(-0.5), batch_statistic_.mutable_gpu_data());
  // Broadcast the inverse std
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1,
      Dtype(1), batch_sum_multiplier_.gpu_data(), batch_statistic_.gpu_data(),
      Dtype(0), spatial_statistic_.mutable_gpu_data());
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_,
      height_ * width_, 1, Dtype(1),
      spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(),
      Dtype(0), broadcast_buffer_.mutable_gpu_data());
  // Multiply with the inverse std
  caffe_gpu_mul(broadcast_buffer_.count(), const_top_data,
      broadcast_buffer_.gpu_data(), top_data);


  // Save the normalized inputs and std for backprop
  if (!frozen_) {
    caffe_copy(broadcast_buffer_.count(), const_top_data,
        x_norm_.mutable_gpu_data());
    caffe_copy(batch_statistic_.count(), batch_statistic_.gpu_data(),
        x_inv_std_.mutable_gpu_data());
  }

  // Scale
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1,
      Dtype(1), batch_sum_multiplier_.gpu_data(), scale_data,
      Dtype(0), spatial_statistic_.mutable_gpu_data());
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_,
      height_ * width_, 1, Dtype(1),
      spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(),
      Dtype(0), broadcast_buffer_.mutable_gpu_data());
  caffe_gpu_mul(broadcast_buffer_.count(), const_top_data,
      broadcast_buffer_.gpu_data(), top_data);

  // Shift
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1,
      Dtype(1), batch_sum_multiplier_.gpu_data(), shift_data,
      Dtype(0), spatial_statistic_.mutable_gpu_data());
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_,
      height_ * width_, 1, Dtype(1),
      spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(),
      Dtype(0), broadcast_buffer_.mutable_gpu_data());
  caffe_gpu_add(broadcast_buffer_.count(), const_top_data,
      broadcast_buffer_.gpu_data(), top_data);


  if (!frozen_ && this->phase_ != TEST && rebn_)
  {
    for (int i=0; i<channels_; i++)
    {
      Dtype s1 = d_.cpu_data()[i] - this->blobs_[2]->cpu_data()[i] , s2 = sqrt(this->blobs_[3]->cpu_data()[i] + bn_eps_);

      if (s2 * this->max_d_ <= s1)
        this->d_.mutable_cpu_data()[i] = this->max_d_;
      else if (-s2 * this->max_d_ >= s1)
        this->d_.mutable_cpu_data()[i] = -this->max_d_;
      else
        this->d_.mutable_cpu_data()[i] = s1 / s2;


      s1 = sqrt(r_.cpu_data()[i] + bn_eps_) , s2 = sqrt(this->blobs_[3]->cpu_data()[i] + bn_eps_);

      if (s2 * this->max_r_ <= s1)
        this->r_.mutable_cpu_data()[i] = this->max_r_;
      else if (s2 / this->max_r_ >= s1)
        this->r_.mutable_cpu_data()[i] = Dtype(1)/this->max_r_;
      else
        this->r_.mutable_cpu_data()[i] = s1 / s2;

      #ifdef USE_MPI
      static int cnt = 0;
      if (++cnt == 400000 && Caffe::MPI_my_rank() == 1)
      {
        cnt = 0;
        LOG(ERROR)  << "ddddddddddddd" << ' '<<  s1 << ' '<< s2 << ' '<< s1/s2 << ' ' << -this->max_d_ << ' ' << this->max_d_ << ' ' << this->d_.cpu_data()[i];
        LOG(ERROR)  << "rrrrrrrrrrrrr" << ' '<<  s1 << ' '<< s2 << ' '<<s1/s2 << ' ' << 1/this->max_r_ << ' ' << this->max_r_ << ' ' << this->r_.cpu_data()[i];
      }
      #endif
    }
    // Broadcast the r
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1,
          Dtype(1), batch_sum_multiplier_.gpu_data(), r_.gpu_data(),
          Dtype(0), spatial_statistic_.mutable_gpu_data());
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_,
        height_ * width_, 1, Dtype(1),
        spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(),
        Dtype(0), broadcast_buffer_.mutable_gpu_data());
    // Multiply r
    caffe_gpu_mul(broadcast_buffer_.count(), const_top_data,
        broadcast_buffer_.gpu_data(), top_data);


    // Broadcast the d
   caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1,
          Dtype(1), batch_sum_multiplier_.gpu_data(), d_.gpu_data(),
          Dtype(0), spatial_statistic_.mutable_gpu_data());
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_,
        height_ * width_, 1, Dtype(1),
        spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(),
        Dtype(0), broadcast_buffer_.mutable_gpu_data());
    // Add d
    caffe_gpu_add(broadcast_buffer_.count(), const_top_data,
        broadcast_buffer_.gpu_data(), top_data); 
  }
}

template <typename Dtype>
void BNLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (frozen_) {
    if (propagate_down[0]) {
      const Dtype* const_top_diff = top[0]->gpu_diff();
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
      // Use the moving average variance
      caffe_copy(batch_statistic_.count(), this->blobs_[3]->gpu_data(),
          batch_statistic_.mutable_gpu_data());
      caffe_gpu_add_scalar(batch_statistic_.count(), bn_eps_,
          batch_statistic_.mutable_gpu_data());
      caffe_gpu_powx(batch_statistic_.count(), batch_statistic_.gpu_data(),
          Dtype(-0.5), batch_statistic_.mutable_gpu_data());
      // Multiple slope with inverse std
      caffe_gpu_mul(batch_statistic_.count(), this->blobs_[0]->gpu_data(),
          batch_statistic_.gpu_data(), batch_statistic_.mutable_gpu_data());
      // Broadcast
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1,
          Dtype(1), batch_sum_multiplier_.gpu_data(), batch_statistic_.gpu_data(),
          Dtype(0), spatial_statistic_.mutable_gpu_data());
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_,
          height_ * width_, 1, Dtype(1),
          spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(),
          Dtype(0), broadcast_buffer_.mutable_gpu_data());
      // Elementwise multiply top grad with (slope / std)
      caffe_gpu_mul(broadcast_buffer_.count(), const_top_diff,
          broadcast_buffer_.gpu_data(), bottom_diff);
    }
    return;
  }

  const Dtype* const_top_diff = top[0]->gpu_diff();
  
  if (rebn_)
  {
    // Broadcast the r
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1,
          Dtype(1), batch_sum_multiplier_.gpu_data(), r_.gpu_data(),
          Dtype(0), spatial_statistic_.mutable_gpu_data());
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_,
        height_ * width_, 1, Dtype(1),
        spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(),
        Dtype(0), broadcast_buffer_.mutable_gpu_data());
    // Multiply r
    caffe_gpu_mul(broadcast_buffer_.count(), const_top_diff,
        broadcast_buffer_.gpu_data(), broadcast_buffer_.mutable_gpu_diff());

    const_top_diff = broadcast_buffer_.gpu_diff();
  }

  // gradient w.r.t. slope
  if (this->param_propagate_down_[0]) {
    
    Dtype* scale_diff = this->blobs_[0]->mutable_gpu_diff();
    caffe_gpu_mul(broadcast_buffer_.count(), x_norm_.gpu_data(), const_top_diff,
        broadcast_buffer_.mutable_gpu_data());
    caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * channels_, height_ * width_,
        Dtype(1), broadcast_buffer_.gpu_data(),
        spatial_sum_multiplier_.gpu_data(), Dtype(0),
        spatial_statistic_.mutable_gpu_data());
    caffe_gpu_gemv<Dtype>(CblasTrans, num_, channels_, Dtype(1),
        spatial_statistic_.gpu_data(), batch_sum_multiplier_.gpu_data(),
        Dtype(1), scale_diff);
  }

  // gradient w.r.t. bias
  if (this->param_propagate_down_[1]) {
    Dtype* shift_diff = this->blobs_[1]->mutable_gpu_diff();
    caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * channels_, height_ * width_,
        Dtype(1), const_top_diff, spatial_sum_multiplier_.gpu_data(),
        Dtype(0), spatial_statistic_.mutable_gpu_data());
    caffe_gpu_gemv<Dtype>(CblasTrans, num_, channels_, Dtype(1),
        spatial_statistic_.gpu_data(), batch_sum_multiplier_.gpu_data(),
        Dtype(1), shift_diff);
  }

  // gradient w.r.t. normalized inputs
  if (propagate_down[0]) {
    const Dtype* const_bottom_diff = bottom[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* scale_data = this->blobs_[0]->gpu_data();
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1,
        Dtype(1), batch_sum_multiplier_.gpu_data(), scale_data,
        Dtype(0), spatial_statistic_.mutable_gpu_data());
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_,
        height_ * width_, 1, Dtype(1), spatial_statistic_.gpu_data(),
        spatial_sum_multiplier_.gpu_data(), Dtype(0),
        broadcast_buffer_.mutable_gpu_data());
    caffe_gpu_mul(broadcast_buffer_.count(), const_top_diff,
        broadcast_buffer_.gpu_data(), broadcast_buffer_.mutable_gpu_data());

    // sum of x_hat * (dl / dx_hat)
    caffe_gpu_mul(broadcast_buffer_.count(), x_norm_.gpu_data(),
        broadcast_buffer_.gpu_data(), bottom_diff);
    caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * channels_, height_ * width_,
        Dtype(1), const_bottom_diff, spatial_sum_multiplier_.gpu_data(),
        Dtype(0), spatial_statistic_.mutable_gpu_data());
    caffe_gpu_gemv<Dtype>(CblasTrans, num_, channels_, Dtype(1),
        spatial_statistic_.gpu_data(), batch_sum_multiplier_.gpu_data(),
        Dtype(0), batch_statistic_.mutable_gpu_data());

    // x_hat times the sum
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1,
        Dtype(1), batch_sum_multiplier_.gpu_data(), batch_statistic_.gpu_data(),
        Dtype(0), spatial_statistic_.mutable_gpu_data());
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_,
        height_ * width_, 1, Dtype(1),
        spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(),
        Dtype(0), bottom_diff);
    caffe_gpu_mul(broadcast_buffer_.count(), x_norm_.gpu_data(),
        const_bottom_diff, bottom_diff);

    // Subtract the average of x_hat times the sum
    caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * channels_, height_ * width_,
        Dtype(1), broadcast_buffer_.gpu_data(),
        spatial_sum_multiplier_.gpu_data(), Dtype(0),
        spatial_statistic_.mutable_gpu_data());
    caffe_gpu_gemv<Dtype>(CblasTrans, num_, channels_, Dtype(1),
        spatial_statistic_.gpu_data(), batch_sum_multiplier_.gpu_data(),
        Dtype(0), batch_statistic_.mutable_gpu_data());
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1,
        Dtype(1), batch_sum_multiplier_.gpu_data(), batch_statistic_.gpu_data(),
        Dtype(0), spatial_statistic_.mutable_gpu_data());
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_,
        height_ * width_, 1, Dtype(1),
        spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(),
        Dtype(1), bottom_diff);
    caffe_gpu_axpby(broadcast_buffer_.count(), Dtype(1),
        broadcast_buffer_.gpu_data(), Dtype(-1) / (num_ * height_ * width_),
        bottom_diff);

    // Multiply with the inverse std
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1,
        Dtype(1), batch_sum_multiplier_.gpu_data(), x_inv_std_.gpu_data(),
        Dtype(0), spatial_statistic_.mutable_gpu_data());
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_,
        height_ * width_, 1, Dtype(1),
        spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(),
        Dtype(0), broadcast_buffer_.mutable_gpu_data());
    caffe_gpu_mul(broadcast_buffer_.count(), const_bottom_diff,
        broadcast_buffer_.gpu_data(), bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(BNLayer);

}  // namespace caffe
